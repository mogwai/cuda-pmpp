#include "hip/hip_runtime.h"
// Compute vecotr Sum C = A + B
// Each thread performs one pair-wise addition
// callable from host executed on device
__global__
void vecAddKernel(float* A, float* B, float* C, int n) {
	// This variable i will be local to the thread

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	// The sequence may not be divisible by the block size:
	// So that means that we need to skip anything bigger than n
	// even if a thread was launched for it.
	if (i < n ) {
		C[i] = A[i] + B[i];
	}
}

void blur(float *A, float* B, float* C, int n ) {
	float *A_d, *B_d, *C_d;
	int size = n * sizeof(float);
	
	// Set up memory on device
	hipMalloc((void **) &A_d, size);
	hipMalloc((void **) &B_d, size);
	hipMalloc((void **) &C_d, size);

	// Copy the vectors to device
	hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
	hipMemcpy(C_d, C, size, hipMemcpyHostToDevice);

	// Execute the Kernel
	// 256 defines the block size and how many threads blocks we'll launch
	// e.g. if n = 1000 we'll launch 4 thread blocks.
	// Execution order will be random as fuck!
	vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);

	// Copy the answer back to host memory
	hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

	// Free up the memory on the device
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
}

int main(int argc, char** argv) {
    // Define vector size
    int n = 1000;

    // Allocate and initialize vectors A, B, and C
    float *A = (float*)malloc(n * sizeof(float));
    float *B = (float*)malloc(n * sizeof(float));
    float *C = (float*)malloc(n * sizeof(float));

    // Initialize vectors A and B with some values
    for (int i = 0; i < n; ++i) {
        A[i] = 1.0f; // Set some arbitrary values for demonstration
        B[i] = 2.0f;
    }

    // Call the vecadd function to perform vector addition on the GPU
    vecadd(A, B, C, n);

    // Free allocated memory
    free(A);
    free(B);
    free(C);

    return 0;
}
